#include "hip/hip_runtime.h"
// Including stuff - should be moved out to iostream
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "profiler.h"


// 10 threads, 1 block 
__global__ void vecAddKernel(float *A, float *B, float *res, int n) {
	
	// Block Index - blockIdx.x
	// Thread Index - threadIdx.x
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < n) {
		res[i] = A[i] + B[i];
	}

}


// THis will be run on your CPU
void vecAdd(float *A, float *B, float *res, int n) {
	float *A_d, *B_d, *res_d;
	size_t size = n * sizeof(float);

	printf("Allocating");
	hipMalloc((void **)&A_d, size);
	hipMalloc((void **)&B_d, size);
	hipMalloc((void **)&res_d, size);

	printf(" Done Allocating, now copying");	
	hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);
	printf(" Done Copying , now setting config params");

	const unsigned int numThreads = 32;
	unsigned int numBlocks = 2;

	printf("Kernel time");
	vecAddKernel<<<numBlocks, numThreads>>>(A_d, B_d, res_d, n);

	hipDeviceSynchronize();

	// Once the exec is done, we move it back from device to host
	hipMemcpy(res, res_d, size, hipMemcpyDeviceToHost);
	printf("haha done copying results?");

	hipFree(A_d);
	hipFree(B_d);
	hipFree(res_d);
	printf("Done freeing mem from device?");
}


// Main fn will include InitTrace() and FinitTrace()
int main() {
	const int n = 16;

	float A[n];
	float B[n];
	float res[n];

	printf("Entered the main fn?");
	for (int i = 0; i < n; i += 1) {
		A[i] = float(i);
		B[i] = A[i] / 1000.0f;
	}

    InitTrace();     // Begin profiling
	vecAdd(A, B, res, n);
	printf("Done with vecadd, bitches");
    finitTrace();    // End profiling
	return 0;
}

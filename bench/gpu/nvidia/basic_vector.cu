#include "hip/hip_runtime.h"
// Including stuff - should be moved out to iostream
#include <hip/hip_runtime.h>
#include <iostream>

// How to include this -- idk
#include "profiler/gpu/profiler.h"

__global__ void vecAddKernel(float *A, float *B, float *res, int n) {
	
	// Block Index - blockIdx.x
	// Thread Index - threadIdx.x
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < n) {
		res[i] = A[i] + B[i];
	}

}



// THis will be run on your CPU
void vecAdd(float *A, float *B, float *res, int n) {
	float *A_d, *B_d, *res_d;
	size_t size = n * sizeof(float);

	hipMalloc((void **)&A_d, size);
	hipMalloc((void **)&B_d, size);
	hipMalloc((void **)&res_d, size);

	hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

	const unsigned int numThreads = 32;
	unsigned int numBlocks = 2;

	vecAddKernel<<<numBlocks, numThreads>>>(A_d, B_d, res_d, n);

    // Tell the CPU that the kernel has finished and we can copy things back
	hipDeviceSynchronize();

	// Once the exec is done, we move it back from device to host
	hipMemcpy(res, res_d, size, hipMemcpyDeviceToHost);

	hipFree(A_d);
	hipFree(B_d);
	hipFree(res_d);
}


// Main fn will include InitTrace() and FinitTrace()
int main() {
	const int n = 16;

	float *A = new float[n];
	float *B = new float[n];
	float *res = new float[n];

	for (int i = 0; i < n; i += 1) {
		A[i] = float(i);
		B[i] = A[i] / 1000.0f;
	}

    InitTrace();     // Begin profiling
	vecAdd(&A, &B, &res, n);
    finitTrace();    // End profiling
	return 0;
}
